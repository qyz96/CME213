#include "hip/hip_runtime.h"
#include "gpu_func.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <iostream>
#include <cassert>
#include <math.h>
#include "hipblas.h"
#define BLOCK_SIZE 32
#define BLOCK_SIZE_X 4
#define BLOCK_SIZE_Y 16
__global__
void device_add_one(int* d_result, int t) {
    *d_result = t + 1;
}

/*
Just a dummy function that can be used to warm up GPU
*/
int useless_gpu_add_one(int t) {
    int result;
    int* d_result;

    checkCudaErrors(hipMalloc((void**)&d_result, 1 * sizeof(int)));

    event_pair timer;
    start_timer(&timer);
    device_add_one<<<1,1>>>(d_result, t);
    check_launch("device_add_one");
    double time = stop_timer(&timer);

    std::cout << "device_add_one took: " << time << " seconds" << std::endl;

    checkCudaErrors(hipMemcpy(&result, d_result, 1 * sizeof(int),
                               hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(d_result));
    return result;
}


__global__
void device_gemm(double* __restrict__ A, double* __restrict__ B,
           double* __restrict__ C, double alpha, double beta,
           int M, int N, int K) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    printf("device_gemm is called!\n");
    if ((i < M) && (j < N)) {
        double temp=0;
        for (int k=0; k<K; k++) {
            temp+=A[i+k*M]*B[k+j*K];
            if ((i==0) && (j==0)) {

                if (k<=5) {printf("w[%d,%d]=%f\n", i, k, A[i+k*M]);}
            }
        }
        C[i+j*M]=alpha*temp+beta*C[i+j*M];
    }
}


__global__
void device_gemm_shared(double* __restrict__ A, double* __restrict__ B,
           double* __restrict__ C, double alpha, double beta,
           int M, int N, int K) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int rj = threadIdx.x;
    int ri = threadIdx.y;
    double temp=0;
    __shared__ double As[BLOCK_SIZE*BLOCK_SIZE];
    __shared__ double Bs[BLOCK_SIZE*BLOCK_SIZE];

    
    int nb = (K+BLOCK_SIZE-1)/BLOCK_SIZE;
    for (int m=0; m<nb; m++)   {
        if ((i<M) && ((BLOCK_SIZE*m+rj)<K)){
            As[ri+BLOCK_SIZE*rj]=A[i+M*(BLOCK_SIZE*m+rj)];
        }
        if ((j<N) && ((BLOCK_SIZE*m+ri)<K)) {
            Bs[ri+BLOCK_SIZE*rj]=B[BLOCK_SIZE*m+ri+K*j];
        }
        __syncthreads();
        if ((i<M) && (j<N)) {
            for (int k=0; k < BLOCK_SIZE; k++) {
                if ((BLOCK_SIZE*m+k) >= K)  {
                    break;
                }
                temp+=As[ri+BLOCK_SIZE*k]*Bs[k+BLOCK_SIZE*rj];
                //printf("Ctrue(%d,%d, %d)+= %f * %f\n", i, k, j, As[ri+BLOCK_SIZE*k], Bs[k+BLOCK_SIZE*rj]);
                
            }
        }
        __syncthreads();
    }
    if ((i<M) && (j<N)) {
            C[i+j*M]=alpha*temp+beta*C[i+j*M];
            //printf("Ctrue(%d,%d)=%f\n", i, j, C[i+j*M]);
        }
    
}

__global__
void device_gemm_shared2(double* __restrict__ A, double* __restrict__ B,
           double* __restrict__ C, double alpha, double beta,
           int M, int N, int K) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int rj = threadIdx.x;
    int ri = threadIdx.y;
    int row = ri + BLOCK_SIZE_Y * rj;
    int i = blockIdx.y * BLOCK_SIZE_Y * BLOCK_SIZE_X + row;
    __shared__ double Bs[BLOCK_SIZE_X*BLOCK_SIZE_Y];

    double As[BLOCK_SIZE_Y];
    double temp[BLOCK_SIZE_X]={0};

    int nb = (K+BLOCK_SIZE_Y-1)/BLOCK_SIZE_Y;
    for (int m=0; m<nb; m++)   {
        if (i<M) {
            for (int ii=0; ii<BLOCK_SIZE_Y;ii++) {
                if ((BLOCK_SIZE_Y*m+ii)>=K) {
                    break;
                }
                As[ii]=A[i+M*(BLOCK_SIZE_Y*m+ii)];
            }
        }
        if ((j<N) && ((BLOCK_SIZE_Y*m+ri)<K)) {
            Bs[ri+BLOCK_SIZE_Y*rj]=B[BLOCK_SIZE_Y*m+ri+K*j];
        }
        __syncthreads();
        if ((i<M)) {
            for (int ii=0; ii<BLOCK_SIZE_X; ii++) {
                if ((blockIdx.x * blockDim.x+ii) >=N) {
                    break;
                }
                for (int k=0; k < BLOCK_SIZE_Y; k++) {
                    if ((BLOCK_SIZE_Y*m+k) >= K)  {
                        break;
                    }
                    temp[ii]+=As[k]*Bs[k+BLOCK_SIZE_Y*ii];
                    
            }
            }
        }
        __syncthreads();
    }
     if ((i<M)) {
            for (int ii=0; ii<BLOCK_SIZE_X; ii++) {
                if ((blockIdx.x * blockDim.x+ii) >=N) {
                    break;
                }
                C[i+M*(blockIdx.x * blockDim.x+ii)]=alpha*temp[ii]+beta*C[i+M*(blockIdx.x * blockDim.x+ii)];
            }
        }
}

__global__
void device_gemm_shared3(double* __restrict__ A, double* __restrict__ B,
           double* __restrict__ C, double alpha, double beta,
           int M, int N, int K) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int rj = threadIdx.x;
    int ri = threadIdx.y;
    //int col = ri + BLOCK_SIZE_Y * rj;
    int col = ri * BLOCK_SIZE_X + rj;
    int j = blockIdx.x * BLOCK_SIZE_X * BLOCK_SIZE_Y + col;
    __shared__ double As[BLOCK_SIZE_X*BLOCK_SIZE_Y];

    double Bs[BLOCK_SIZE_X];
    double temp[BLOCK_SIZE_Y]={0};
    int nb = (K+BLOCK_SIZE_X-1)/BLOCK_SIZE_X;
    for (int m=0; m<nb; m++)   {

        if (j<N) {
            for (int ii=0; ii<BLOCK_SIZE_X;ii++) {
                if ((ii+BLOCK_SIZE_X*m)>=K) {
                    break;
                }
                Bs[ii]=B[ii+BLOCK_SIZE_X*m+K*j];
                //printf("Bs[%d]=B(%d, %d)=%f\n", ii, ii+BLOCK_SIZE_X*m, j, B[ii+BLOCK_SIZE_X*m+N*j]);
            }
        }


        if ((i<M) && ((BLOCK_SIZE_X*m+rj)<K)) {
            As[ri+BLOCK_SIZE_Y*rj]=A[i+M*(rj+BLOCK_SIZE_X*m)];
            //printf("A(%d, %d)=%f\n", i, rj+BLOCK_SIZE_X*m, A[i+M*(rj+BLOCK_SIZE_X*m)]);
        }


        __syncthreads();

        if ((j<N)) {
            for (int ii=0; ii<BLOCK_SIZE_Y; ii++) {
                if ((blockIdx.y * blockDim.y+ii) >=M) {
                    break;
                }
                for (int k=0; k < BLOCK_SIZE_X; k++) {
                    if ((BLOCK_SIZE_X*m+k) >= K)  {
                        break;
                    }
                    temp[ii]+=As[ii+BLOCK_SIZE_Y*k]*Bs[k];
                    //printf("C(%d, %d, %d)+= %f * %f\n", blockIdx.y * blockDim.y+ii, BLOCK_SIZE_X*m+k, j, As[ii+BLOCK_SIZE_Y*k],Bs[k]);
            }
            }
        }
        __syncthreads();
    }

    if ((j<N)) {
        for (int ii=0; ii<BLOCK_SIZE_Y; ii++) {
            if ((blockIdx.y * blockDim.y+ii) >=M) {
                break;
            }
            C[blockIdx.y * blockDim.y+ii+M*j]=alpha*temp[ii]+beta*C[blockIdx.y * blockDim.y+ii+M*j];
            //printf("C(%d,%d)=%f\n", blockIdx.y * blockDim.y+ii, j, C[blockIdx.y * blockDim.y+ii+M*j]);
        }
    }
}

/*
Routine to perform an in-place GEMM operation, i.e., C := alpha*A*B + beta*C
*/
int myGEMM(double* __restrict__ A, double* __restrict__ B,
           double* __restrict__ C, double* alpha, double* beta,
           int M, int N, int K) {
    /* TODO: Write an efficient GEMM implementation on GPU */
    double al=*alpha;
    double be=*beta;
    
    int block_size_x = BLOCK_SIZE;
    int block_size_y = BLOCK_SIZE;
    int numBlocks_x = (N + block_size_x - 1) / block_size_x;
    int numBlocks_y = (M + block_size_y - 1) / (block_size_y);
    
/*     int block_size_x = BLOCK_SIZE_X;
    int block_size_y = BLOCK_SIZE_Y;
    int numBlocks_x = (N + block_size_x * block_size_y  - 1) / (block_size_y * block_size_x);
    int numBlocks_x = (N + block_size_x - 1) / (block_size_x);
    int numBlocks_y = (M + block_size_y - 1) / (block_size_y); */
    printf("myGEMM is called!\n");
    dim3 threads(block_size_x, block_size_y);
    dim3 blocks(numBlocks_x, numBlocks_y);
    device_gemm<<<blocks, threads>>>(A, B, C, al, be, M, N, K);
    /*
    block_size_x = BLOCK_SIZE;
    block_size_y = BLOCK_SIZE;
    numBlocks_x = (N + block_size_x - 1) / block_size_x;
    numBlocks_y = (M + block_size_y - 1) / (block_size_y);
    dim3 threads1(block_size_x, block_size_y);
    dim3 blocks1(numBlocks_x, numBlocks_y);
    device_gemm_shared<<<blocks1, threads1>>>(A, B, C, al, be, M, N, K);
    */
    
    return 0;
}


/*
Compute C = alpha * A + beta * B
*/

__global__
void device_addmat(double* A, double* B, double* C, double alpha, double beta, int M, int N) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if ((i < M) && (j < N)) {
        C[i + j * M] = alpha * A[i + j * M] + beta * B[i + j * M];
    }
    return;

}


void gpu_addmat(double* dA, double* dB, double* dC, double alpha, double beta, int M, int N)  {

    int block_size_x = BLOCK_SIZE;
    int block_size_y = BLOCK_SIZE;
    int numBlocks_x = (N + block_size_x - 1) / block_size_x;
    int numBlocks_y = (M + block_size_y - 1) / (block_size_y);
    dim3 threads(block_size_x, block_size_y);
    dim3 blocks(numBlocks_x, numBlocks_y);
    device_addmat<<<blocks, threads>>>(dA, dB, dC, alpha, beta, M, N);
}


__global__
void device_sumcol(double* data, double* result, int M, int N) {


    extern __shared__ double sdata[];
    int i = threadIdx.y;
    int j = blockIdx.x;
    if (j < N) {
        if (i < M) {
            sdata[i] = data[i + j * M];
        }
        else {
            sdata[i] = 0;
        }
        __syncthreads();
        for (unsigned int s=0; s < blockDim.y; s *= 2) {
            int index = 2 * s * i;
            if (index < blockDim.y) {
                sdata[index] += sdata[index+s];
            }
        }
        __syncthreads();


        result[j]=sdata[0];
    }
}

void gpu_sumcol(double* ddata, double* dresult, int M, int N) {

    int block_size_x = 1;
    int block_size_y = 16;
    int numBlocks_x = (N + block_size_x - 1) / block_size_x;
    int numBlocks_y = (M + block_size_y - 1) / (block_size_y);
    dim3 threads(block_size_x, block_size_y);
    dim3 blocks(numBlocks_x, numBlocks_y);
    device_sumcol<<<blocks, threads>>>(ddata, dresult, M, N);
}

__global__
void device_transpose(double* data, double* result, int M, int N)  {

    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if ((i < M) && (j < N)) {
        result[j + i * N] = data[i + j * M];
    }
    return; 
}

void gpu_transpose(double* ddata, double* dresult, int M, int N)  {

    int block_size_x = 32;
    int block_size_y = 32;
    int numBlocks_x = (N + block_size_x - 1) / block_size_x;
    int numBlocks_y = (M + block_size_y - 1) / (block_size_y);
    dim3 threads(block_size_x, block_size_y);
    dim3 blocks(numBlocks_x, numBlocks_y);

    device_transpose<<<blocks, threads>>>(ddata, dresult, M, N);
}


__global__
void device_sigmoid(double* data, double* result, int M, int N) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if ((i < M) && (j < N)) {
        result[i + j * M] = 1 / (double)(1+std::exp(-data[i + j * M]));
    }
    return;
}


void gpu_sigmoid(double* ddata, double* dresult, int M, int N)  {

    int block_size_x = BLOCK_SIZE;
    int block_size_y = BLOCK_SIZE;
    int numBlocks_x = (N + block_size_x - 1) / block_size_x;
    int numBlocks_y = (M + block_size_y - 1) / (block_size_y);
    dim3 threads(block_size_x, block_size_y);
    dim3 blocks(numBlocks_x, numBlocks_y);

    device_sigmoid<<<blocks, threads>>>(ddata, dresult, M, N);
}





__global__
void device_exp(double* data, double* result, int M, int N) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if ((i < M) && (j < N)) {
        result[i + j * M] = (double)(std::exp(data[i + j * M]));
    }
    return;
}


void gpu_exp(double* ddata, double* dresult, int M, int N) {

    int block_size_x = BLOCK_SIZE;
    int block_size_y = BLOCK_SIZE;
    int numBlocks_x = (N + block_size_x - 1) / block_size_x;
    int numBlocks_y = (M + block_size_y - 1) / (block_size_y);
    dim3 threads(block_size_x, block_size_y);
    dim3 blocks(numBlocks_x, numBlocks_y);

    device_sigmoid<<<blocks, threads>>>(ddata, dresult, M, N);
}


__global__
void device_softmax(double* data, double* result, int M, int N) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if ((i < M) && (j < N)) {
        result[i + j * M] = (double)((result[i + j * M])/(data[j]));
    }
    return;
}


void gpu_softmax(double* ddata, double* dresult, int M, int N)  {
    int block_size_x = BLOCK_SIZE;
    int block_size_y = BLOCK_SIZE;
    int numBlocks_x = (N + block_size_x - 1) / block_size_x;
    int numBlocks_y = (M + block_size_y - 1) / (block_size_y);
    dim3 threads(block_size_x, block_size_y);
    dim3 blocks(numBlocks_x, numBlocks_y);

    device_softmax<<<blocks, threads>>>(ddata, dresult, M, N);


}


__global__
void device_hadmard(double* c, double* a, double* b, int M, int N) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if ((i < M) && (j < N)) {
        c[i + j * M] = (double)((a[i + j * M]) * (b[i + j * M]) * (1 - b[i + j * M]));
    }
    return;
}


void gpu_hadmard(double* c, double* a, double* b, int M, int N) {

    int block_size_x = BLOCK_SIZE;
    int block_size_y = BLOCK_SIZE;
    int numBlocks_x = (N + block_size_x - 1) / block_size_x;
    int numBlocks_y = (M + block_size_y - 1) / (block_size_y);
    dim3 threads(block_size_x, block_size_y);
    dim3 blocks(numBlocks_x, numBlocks_y);

    device_hadmard<<<blocks, threads>>>(c, a, b, M, N);
}




void my_feedforward(NeuralNetwork& nn, const arma::mat& X, struct cache& cache, 
    const arma::mat& b0r, const arma::mat& b1r, const arma::mat& T, double* a0, double* a1, double* z0, double* z1, double* yc, double* W1_test, double* W0_test, double* W3_test) {

    double* dz0;
    double* dz1;
    double* da0;
    double* da1;
    double* dW0;
    double* dW1;
    double* db0;
    double* db1;
    double* dX;
    double* dT;
    double* dexp;

    
    int num_sample = X.n_cols;
    int K = nn.W[0].n_rows;
    int M = nn.W[0].n_cols;
    int N = nn.W[1].n_rows;


    //std::cout<<"Allocating CUDA memory....\n";
    hipMalloc((void**)&dz0, sizeof(double) * K * num_sample);
    hipMalloc((void**)&dz1, sizeof(double) * N * num_sample);
    hipMalloc((void**)&da0, sizeof(double) * K * num_sample);
    hipMalloc((void**)&da1, sizeof(double) * N * num_sample);
    hipMalloc((void**)&dW0, sizeof(double) * M * K);
    hipMalloc((void**)&dW1, sizeof(double) * K * N);
    hipMalloc((void**)&db0, sizeof(double) * K * num_sample);
    hipMalloc((void**)&db1, sizeof(double) * N * num_sample);
    hipMalloc((void**)&dX, sizeof(double) * K * num_sample);
    hipMalloc((void**)&dT, sizeof(double) * N * num_sample);
    hipMalloc((void**)&dexp, sizeof(double) * 1 * num_sample);

    
    //std::cout<<"Copying CUDA memory....\n";
    hipMemcpy(dz0, b0r.memptr(), sizeof(double) * K * num_sample , hipMemcpyHostToDevice);
    hipMemcpy(dz1, b1r.memptr(), sizeof(double) * N * num_sample, hipMemcpyHostToDevice);
    hipMemcpy(da0, a0, sizeof(double) * K * num_sample, hipMemcpyHostToDevice);
    hipMemcpy(da1, a1, sizeof(double) * N * num_sample, hipMemcpyHostToDevice);


    //hipMemcpy(dW0, nn.W[0].memptr(), sizeof(double) * M * K, hipMemcpyHostToDevice);


    hipMemcpy(dW0, nn.W[0].memptr(), sizeof(double) * M * K, hipMemcpyHostToDevice);
    hipMemcpy(dW1, nn.W[1].memptr(), sizeof(double) * K * N, hipMemcpyHostToDevice);
    hipMemcpy(dT, T.memptr(), sizeof(double) * N * num_sample, hipMemcpyHostToDevice);
    hipMemcpy(dX, X.memptr(), sizeof(double) * M * num_sample, hipMemcpyHostToDevice);



    //std::cout<<"nn.W[0] * X + arma::repmat(nn.b[0], 1, N)....\n";
    double alpha = 1;
    double beta = 1;


    myGEMM(dW0, dX, dz0, &alpha, &beta, K, num_sample, M);
    gpu_sigmoid(dz0, da0, K, num_sample);
    std::cout<<"nn.W[0] * X + arma::repmat(nn.b[0], 1, N)....\n";
    myGEMM(dW1, da0, dz1, &alpha, &beta, N, num_sample, K);

    
    gpu_exp(dz1, da1, N, num_sample);
    gpu_sumcol(da1, dexp, N, num_sample);
    gpu_softmax(dexp, da1, N, num_sample);
    hipMemcpy(a1, da1, sizeof(double) * N * num_sample, hipMemcpyDeviceToHost);
    hipMemcpy(yc, da1, sizeof(double) * N * num_sample, hipMemcpyDeviceToHost);
    hipMemcpy(z0, dz0, sizeof(double) * K * num_sample, hipMemcpyDeviceToHost);
    hipMemcpy(a0, da0, sizeof(double) * K * num_sample, hipMemcpyDeviceToHost);
    hipMemcpy(W3_test, dW1, sizeof(double) * N * K, hipMemcpyDeviceToHost);
    hipMemcpy(z1, dz1, sizeof(double) * N * num_sample, hipMemcpyDeviceToHost);



}





void my_backprop(NeuralNetwork& nn, const arma::mat& y, double reg, const struct cache& bpcache, struct grads& bpgrads) {
    int num_sample = bpcache.X.n_cols;
    int K = nn.W[0].n_rows;
    int M = nn.W[0].n_cols;
    int N = nn.W[1].n_rows;
    
    bpgrads.dW.resize(2);
    bpgrads.dW[0].zeros(K, M);
    bpgrads.dW[1].zeros(N, K);
    bpgrads.db.resize(2);
    bpgrads.db[0].zeros(K);
    bpgrads.db[1].zeros(N);

    arma::vec allones = arma::ones<arma::vec>(num_sample);

    double* dW0;
    double* da0;
    double* dW1;
    double* db0;
    double* db1;
    double* dyc;
    double* dy;
    double* dDff;
    double* dOne;
    double* daz;
    double* dX;

    hipMalloc((void**)&dW0, sizeof(double) * M * K);
    hipMalloc((void**)&dW1, sizeof(double) * K * N);
    hipMalloc((void**)&db0, sizeof(double) * K);
    hipMalloc((void**)&db1, sizeof(double) * N);
    hipMalloc((void**)&dyc, sizeof(double) * N * num_sample);
    hipMalloc((void**)&dy, sizeof(double) * N * num_sample);
    hipMalloc((void**)&dOne, sizeof(double) * num_sample);
    hipMalloc((void**)&daz, sizeof(double) * K * num_sample);
    hipMalloc((void**)&dX, sizeof(double) * M * num_sample);
    hipMalloc((void**)&da0, sizeof(double) * K * num_sample);

    hipMemcpy(dW0, nn.W[0].memptr(), sizeof(double) * M * K, hipMemcpyHostToDevice);
    hipMemcpy(dW1, nn.W[1].memptr(), sizeof(double) * K * N, hipMemcpyHostToDevice);
    hipMemcpy(db0, nn.b[0].memptr(), sizeof(double) * K, hipMemcpyHostToDevice);
    hipMemcpy(db1, nn.b[1].memptr(), sizeof(double) * N, hipMemcpyHostToDevice);
    hipMemcpy(dyc, bpcache.yc.memptr(), sizeof(double) * N * num_sample, hipMemcpyHostToDevice);
    hipMemcpy(dy, y.memptr(), sizeof(double) * N * num_sample, hipMemcpyHostToDevice);
    hipMemcpy(dOne, allones.memptr(), sizeof(double) * num_sample, hipMemcpyHostToDevice);
    hipMemcpy(dX, bpcache.X.memptr(), sizeof(double) * M * num_sample, hipMemcpyHostToDevice);
    hipMemcpy(da0, bpcache.a[0].memptr(), sizeof(double) * K * num_sample, hipMemcpyHostToDevice);


    int block_size_x = 32;
    int block_size_y = 32;
    int numBlocks_x = (num_sample + block_size_x - 1) / block_size_x;
    int numBlocks_y = (K + block_size_y - 1) / (block_size_y);
    dim3 threads(block_size_x, block_size_y);
    dim3 blocks(numBlocks_x, numBlocks_y);
    double alpha = 1/(double)(num_sample);
    double beta = -1/(double)(num_sample);
    double alpha1 = 1;
    double beta1=0;
    device_addmat<<<blocks, threads>>>(dyc, dy, dy, alpha, beta, N, num_sample);

    hipError_t cudaStat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    stat = hipblasCreate(&handle);
    /*
    stat = hipblasDgemm(handle,
        HIPBLAS_OP_T, HIPBLAS_OP_N,
        K, num_sample, N,
        &alpha1,
        dW1, K,
        dDff, N,
        &beta1,
        daz, K);

    */
    myGEMM(dW1, dDff, daz, &alpha1, &beta1, K, N, num_sample);


    /*
    stat = hipblasDgemm(handle,
        HIPBLAS_OP_N, HIPBLAS_OP_T,
        N, K, num_sample,
        &alpha1,
        dy, N,
        da0, num_sample,
        &reg,
        dW1, N);

    */

    myGEMM(dy, da0, dW1, &alpha1, &reg, N, num_sample, K);


    /*

    stat = hipblasDgemm(handle,
        HIPBLAS_OP_N, HIPBLAS_OP_N,
        N, 1, num_sample,
        &alpha1,
        dDff, N,
        dOne, num_sample,
        &beta1,
        db1, N);
    */
    
    myGEMM(dDff, dOne, db1, &alpha1, &beta1, N, 1, num_sample);
    device_hadmard<<<blocks, threads>>>(daz, daz, da0, K, num_sample);

    stat = hipblasDgemm(handle,
        HIPBLAS_OP_N, HIPBLAS_OP_T,
        K, M, num_sample,
        &alpha1,
        daz, K,
        dX, num_sample,
        &reg,
        dW0, N);



    myGEMM(daz, dX, dW0, &alpha1, &beta1, N, 1, num_sample);
    stat = hipblasDgemm(handle,
        HIPBLAS_OP_N, HIPBLAS_OP_N,
        K, num_sample, 1,
        &alpha1,
        daz, K,
        dOne, num_sample,
        &beta1,
        db0, K);

    myGEMM(daz, dOne, db0, &alpha1, &beta1, K, num_sample, 1);
    
    //std::cout << "backprop " << bpcache.yc << "\n";


    hipMemcpy(bpgrads.dW[0].memptr(), dW0, sizeof(double) * M * K, hipMemcpyDeviceToHost);
    hipMemcpy(bpgrads.db[0].memptr(), db0, sizeof(double) * K, hipMemcpyDeviceToHost);
    hipMemcpy(bpgrads.dW[1].memptr(), dW1, sizeof(double) * N * K, hipMemcpyDeviceToHost);
    hipMemcpy(bpgrads.db[1].memptr(), db1, sizeof(double) * N, hipMemcpyDeviceToHost);

    /*
    arma::mat diff = (1.0 / N) * (bpcache.yc - y);
    bpgrads.dW[1] = diff * bpcache.a[0].t() + reg * nn.W[1];
    bpgrads.db[1] = arma::sum(diff, 1);
    arma::mat da1 = nn.W[1].t() * diff;

    arma::mat dz1 = da1 % bpcache.a[0] % (1 - bpcache.a[0]);

    bpgrads.dW[0] = dz1 * bpcache.X.t() + reg * nn.W[0];
    bpgrads.db[0] = arma::sum(dz1, 1);

    */


}
