#include "hip/hip_runtime.h"
#include "gpu_func.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <iostream>
#include "hipblas.h"
#define BLOCK_SIZE 32
__global__
void device_add_one(int* d_result, int t) {
    *d_result = t + 1;
}

/*
Just a dummy function that can be used to warm up GPU
*/
int useless_gpu_add_one(int t) {
    int result;
    int* d_result;

    checkCudaErrors(hipMalloc((void**)&d_result, 1 * sizeof(int)));

    event_pair timer;
    start_timer(&timer);
    device_add_one<<<1,1>>>(d_result, t);
    check_launch("device_add_one");
    double time = stop_timer(&timer);

    std::cout << "device_add_one took: " << time << " seconds" << std::endl;

    checkCudaErrors(hipMemcpy(&result, d_result, 1 * sizeof(int),
                               hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(d_result));
    return result;
}


__global__
void device_gemm(double* __restrict__ A, double* __restrict__ B,
           double* __restrict__ C, double alpha, double beta,
           int M, int N, int K) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    if ((i < M) && (j < N)) {
        double temp=0;
        for (int k=0; k<K; k++) {
            temp+=A[i+k*M]*B[k+j*K];
        }
        C[i+j*M]=alpha*temp+beta*C[i+j*M];
    }
}


__global__
void device_gemm_shareds(double* __restrict__ A, double* __restrict__ B,
           double* __restrict__ C, double alpha, double beta,
           int M, int N, int K) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int rj = blockIdx.x;
    int ri = blockIdx.y;
    double temp=0;
    __shared__ float As[BLOCK_SIZE*BLOCK_SIZE];
    __shared__ float Bs[BLOCK_SIZE*BLOCK_SIZE];
    int nb = K/BLOCK_SIZE;
    for (int m=0; m<nb; m++)   {
        if ((i<M) && (j<N)) {
            As[ri+BLOCK_SIZE*rj]=A[i+M*(BLOCK_SIZE*m+rj)];
            Bs[ri+BLOCK_SIZE*rj]=B[BLOCK_SIZE*m+ri+K*j];
        }
        __syncthreads();
        if ((i<M) && (j<N)) {
            for (int k=0; k < BLOCK_SIZE; k++) {
                temp+=As[ri+BLOCK_SIZE*k]*Bs[k+BLOCK_SIZE*rj];
            }
        }
        __syncthreads();
    }
    if ((i<M) && (j<N)) {
            C[i+j*M]=alpha*temp+beta*C[i+j*M];
        }
}

/*
Routine to perform an in-place GEMM operation, i.e., C := alpha*A*B + beta*C
*/
int myGEMM(double* __restrict__ A, double* __restrict__ B,
           double* __restrict__ C, double* alpha, double* beta,
           int M, int N, int K) {
    /* TODO: Write an efficient GEMM implementation on GPU */
    double al=*alpha;
    double be=*beta;
    int block_size_x = 32;
    int block_size_y = 32;
    int numBlocks_x = (N + block_size_x - 1) / block_size_x;
    int numBlocks_y = (M + block_size_y - 1) / (block_size_y);
    dim3 threads(block_size_x, block_size_y);
    dim3 blocks(numBlocks_x, numBlocks_y);
    device_gemm<<<blocks, threads>>>(A, B, C, al, be, M, N, K);
    return 0;
}
